#include "hip/hip_runtime.h"
/* Copyright 2017 NVIDIA Corporation
 *
 * The U.S. Department of Energy funded the development of this software 
 * under subcontract B609478 with Lawrence Livermore National Security, LLC
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "snap_types.h"
#include "accessor.h"
#include "snap_cuda_help.h"

using namespace LegionRuntime::Arrays;
using namespace LegionRuntime::Accessor;

template<int GROUPS>
__global__
void gpu_expand_cross_section(const PointerBuffer<GROUPS,double> sig_ptrs,
                              const int    *mat_ptr,
                                    PointerBuffer<GROUPS,double> xs_ptrs,
                              const ByteOffsetArray<1> sig_offsets,
                              const ByteOffsetArray<3> mat_offsets,
                              const ByteOffsetArray<3> xs_offsets)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int z = blockIdx.z * blockDim.z + threadIdx.z;

  const int mat = *(mat_ptr + x * mat_offsets[0] + 
                              y * mat_offsets[1] + 
                              z * mat_offsets[2]) - 1/*IS starts at 1*/;
  #pragma unroll
  for (int g = 0; g < GROUPS; g++) {
    const double *sig_ptr = sig_ptrs[g] + mat * sig_offsets[0];
    double val;
    asm volatile("ld.global.cs.f64 %0, [%1];" : "=d"(val) : "l"(sig_ptr) : "memory");
    double *xs_ptr = xs_ptrs[g] + x * xs_offsets[0] +
                                  y * xs_offsets[1] + z * xs_offsets[2];
    asm volatile("st.global.cs.f64 [%0], %1;" : : "l"(xs_ptr), "d"(val) : "memory");
  }
}

__host__
void run_expand_cross_section(const std::vector<double*> &sig_ptrs,
                              const int *mat_ptr,
                              const std::vector<double*> &xs_ptrs,
                              const ByteOffset sig_offsets[1],
                              const ByteOffset mat_offsets[3],
                              const ByteOffset xs_offsets[3],
                              const Rect<3> &subgrid_bounds)
{
  // Figure out the dimensions to launch
  const int x_range = (subgrid_bounds.hi[0] - subgrid_bounds.lo[0]) + 1; 
  const int y_range = (subgrid_bounds.hi[1] - subgrid_bounds.lo[1]) + 1;
  const int z_range = (subgrid_bounds.hi[2] - subgrid_bounds.lo[2]) + 1;

  dim3 block(gcd(x_range,32), gcd(y_range,4), gcd(z_range,4));
  dim3 grid(x_range/block.x, y_range/block.y, z_range/block.z);

  // Switch on the number of groups
  assert(sig_ptrs.size() == xs_ptrs.size());
  // TODO: replace this template foolishness with Terra
  switch (sig_ptrs.size())
  {
    case 1:
      {
        gpu_expand_cross_section<1><<<grid, block>>>(
                                       PointerBuffer<1,double>(sig_ptrs), mat_ptr,
                                       PointerBuffer<1,double>(xs_ptrs), 
                                       ByteOffsetArray<1>(sig_offsets),
                                       ByteOffsetArray<3>(mat_offsets),
                                       ByteOffsetArray<3>(xs_offsets));
        break;
      }
    case 2:
      {
        gpu_expand_cross_section<2><<<grid, block>>>(
                                       PointerBuffer<2,double>(sig_ptrs), mat_ptr,
                                       PointerBuffer<2,double>(xs_ptrs), 
                                       ByteOffsetArray<1>(sig_offsets),
                                       ByteOffsetArray<3>(mat_offsets),
                                       ByteOffsetArray<3>(xs_offsets));
        break;
      }
    case 3:
      {
        gpu_expand_cross_section<3><<<grid, block>>>(
                                       PointerBuffer<3,double>(sig_ptrs), mat_ptr,
                                       PointerBuffer<3,double>(xs_ptrs), 
                                       ByteOffsetArray<1>(sig_offsets),
                                       ByteOffsetArray<3>(mat_offsets),
                                       ByteOffsetArray<3>(xs_offsets));
        break;
      }
    case 4:
      {
        gpu_expand_cross_section<4><<<grid, block>>>(
                                       PointerBuffer<4,double>(sig_ptrs), mat_ptr,
                                       PointerBuffer<4,double>(xs_ptrs), 
                                       ByteOffsetArray<1>(sig_offsets),
                                       ByteOffsetArray<3>(mat_offsets),
                                       ByteOffsetArray<3>(xs_offsets));
        break;
      }
    case 5:
      {
        gpu_expand_cross_section<5><<<grid, block>>>(
                                       PointerBuffer<5,double>(sig_ptrs), mat_ptr,
                                       PointerBuffer<5,double>(xs_ptrs), 
                                       ByteOffsetArray<1>(sig_offsets),
                                       ByteOffsetArray<3>(mat_offsets),
                                       ByteOffsetArray<3>(xs_offsets));
        break;
      }
    case 6:
      {
        gpu_expand_cross_section<6><<<grid, block>>>(
                                       PointerBuffer<6,double>(sig_ptrs), mat_ptr,
                                       PointerBuffer<6,double>(xs_ptrs), 
                                       ByteOffsetArray<1>(sig_offsets),
                                       ByteOffsetArray<3>(mat_offsets),
                                       ByteOffsetArray<3>(xs_offsets));
        break;
      }
    case 7:
      {
        gpu_expand_cross_section<7><<<grid, block>>>(
                                       PointerBuffer<7,double>(sig_ptrs), mat_ptr,
                                       PointerBuffer<7,double>(xs_ptrs), 
                                       ByteOffsetArray<1>(sig_offsets),
                                       ByteOffsetArray<3>(mat_offsets),
                                       ByteOffsetArray<3>(xs_offsets));
        break;
      }
    case 8:
      {
        gpu_expand_cross_section<8><<<grid, block>>>(
                                       PointerBuffer<8,double>(sig_ptrs), mat_ptr,
                                       PointerBuffer<8,double>(xs_ptrs), 
                                       ByteOffsetArray<1>(sig_offsets),
                                       ByteOffsetArray<3>(mat_offsets),
                                       ByteOffsetArray<3>(xs_offsets));
        break;
      }
    case 9:
      {
        gpu_expand_cross_section<9><<<grid, block>>>(
                                       PointerBuffer<9,double>(sig_ptrs), mat_ptr,
                                       PointerBuffer<9,double>(xs_ptrs), 
                                       ByteOffsetArray<1>(sig_offsets),
                                       ByteOffsetArray<3>(mat_offsets),
                                       ByteOffsetArray<3>(xs_offsets));
        break;
      }
    case 10:
      {
        gpu_expand_cross_section<10><<<grid, block>>>(
                                       PointerBuffer<10,double>(sig_ptrs), mat_ptr,
                                       PointerBuffer<10,double>(xs_ptrs), 
                                       ByteOffsetArray<1>(sig_offsets),
                                       ByteOffsetArray<3>(mat_offsets),
                                       ByteOffsetArray<3>(xs_offsets));
        break;
      }
    case 11:
      {
        gpu_expand_cross_section<11><<<grid, block>>>(
                                       PointerBuffer<11,double>(sig_ptrs), mat_ptr,
                                       PointerBuffer<11,double>(xs_ptrs), 
                                       ByteOffsetArray<1>(sig_offsets),
                                       ByteOffsetArray<3>(mat_offsets),
                                       ByteOffsetArray<3>(xs_offsets));
        break;
      }
    case 12:
      {
        gpu_expand_cross_section<12><<<grid, block>>>(
                                       PointerBuffer<12,double>(sig_ptrs), mat_ptr,
                                       PointerBuffer<12,double>(xs_ptrs), 
                                       ByteOffsetArray<1>(sig_offsets),
                                       ByteOffsetArray<3>(mat_offsets),
                                       ByteOffsetArray<3>(xs_offsets));
        break;
      }
    case 13:
      {
        gpu_expand_cross_section<13><<<grid, block>>>(
                                       PointerBuffer<13,double>(sig_ptrs), mat_ptr,
                                       PointerBuffer<13,double>(xs_ptrs), 
                                       ByteOffsetArray<1>(sig_offsets),
                                       ByteOffsetArray<3>(mat_offsets),
                                       ByteOffsetArray<3>(xs_offsets));
        break;
      }
    case 14:
      {
        gpu_expand_cross_section<14><<<grid, block>>>(
                                       PointerBuffer<14,double>(sig_ptrs), mat_ptr,
                                       PointerBuffer<14,double>(xs_ptrs), 
                                       ByteOffsetArray<1>(sig_offsets),
                                       ByteOffsetArray<3>(mat_offsets),
                                       ByteOffsetArray<3>(xs_offsets));
        break;
      }
    case 15:
      {
        gpu_expand_cross_section<15><<<grid, block>>>(
                                       PointerBuffer<15,double>(sig_ptrs), mat_ptr,
                                       PointerBuffer<15,double>(xs_ptrs), 
                                       ByteOffsetArray<1>(sig_offsets),
                                       ByteOffsetArray<3>(mat_offsets),
                                       ByteOffsetArray<3>(xs_offsets));
        break;
      }
    case 16:
      {
        gpu_expand_cross_section<16><<<grid, block>>>(
                                       PointerBuffer<16,double>(sig_ptrs), mat_ptr,
                                       PointerBuffer<16,double>(xs_ptrs), 
                                       ByteOffsetArray<1>(sig_offsets),
                                       ByteOffsetArray<3>(mat_offsets),
                                       ByteOffsetArray<3>(xs_offsets));
        break;
      }
    case 24:
      {
        gpu_expand_cross_section<24><<<grid, block>>>(
                                       PointerBuffer<24,double>(sig_ptrs), mat_ptr,
                                       PointerBuffer<24,double>(xs_ptrs), 
                                       ByteOffsetArray<1>(sig_offsets),
                                       ByteOffsetArray<3>(mat_offsets),
                                       ByteOffsetArray<3>(xs_offsets));
        break;
      }
    case 32:
      {
        gpu_expand_cross_section<32><<<grid, block>>>(
                                       PointerBuffer<32,double>(sig_ptrs), mat_ptr,
                                       PointerBuffer<32,double>(xs_ptrs), 
                                       ByteOffsetArray<1>(sig_offsets),
                                       ByteOffsetArray<3>(mat_offsets),
                                       ByteOffsetArray<3>(xs_offsets));
        break;
      }
    case 40:
      {
        gpu_expand_cross_section<40><<<grid, block>>>(
                                       PointerBuffer<40,double>(sig_ptrs), mat_ptr,
                                       PointerBuffer<40,double>(xs_ptrs), 
                                       ByteOffsetArray<1>(sig_offsets),
                                       ByteOffsetArray<3>(mat_offsets),
                                       ByteOffsetArray<3>(xs_offsets));
        break;
      }
    case 48:
      {
        gpu_expand_cross_section<48><<<grid, block>>>(
                                       PointerBuffer<48,double>(sig_ptrs), mat_ptr,
                                       PointerBuffer<48,double>(xs_ptrs), 
                                       ByteOffsetArray<1>(sig_offsets),
                                       ByteOffsetArray<3>(mat_offsets),
                                       ByteOffsetArray<3>(xs_offsets));
        break;
      }
    case 56:
      {
        gpu_expand_cross_section<56><<<grid, block>>>(
                                       PointerBuffer<56,double>(sig_ptrs), mat_ptr,
                                       PointerBuffer<56,double>(xs_ptrs), 
                                       ByteOffsetArray<1>(sig_offsets),
                                       ByteOffsetArray<3>(mat_offsets),
                                       ByteOffsetArray<3>(xs_offsets));
        break;
      }
    case 64:
      {
        gpu_expand_cross_section<64><<<grid, block>>>(
                                       PointerBuffer<64,double>(sig_ptrs), mat_ptr,
                                       PointerBuffer<64,double>(xs_ptrs), 
                                       ByteOffsetArray<1>(sig_offsets),
                                       ByteOffsetArray<3>(mat_offsets),
                                       ByteOffsetArray<3>(xs_offsets));
        break;
      }
    default:
      assert(false); // add more cases
  }
}

template<int GROUPS>
__global__
void gpu_expand_scattering_cross_section(const PointerBuffer<GROUPS,MomentQuad> slgg_ptrs,
                                         const int        *mat_ptr,
                                               PointerBuffer<GROUPS,MomentQuad> xs_ptrs,
                                         const ByteOffsetArray<2> slgg_offsets,
                                         const ByteOffsetArray<3> mat_offsets,
                                         const ByteOffsetArray<3> xs_offsets,
                                         const int group_start)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int z = blockIdx.z * blockDim.z + threadIdx.z;

  const int mat = *(mat_ptr + x * mat_offsets[0] + 
                              y * mat_offsets[1] + 
                              z * mat_offsets[2]) - 1/*IS starts at 1*/;
  #pragma unroll
  for (int g = 0; g < GROUPS; g++) {
    MomentQuad quad = *(slgg_ptrs[g] + mat * slgg_offsets[0] +
                        (group_start + g) * slgg_offsets[1]);
    *(xs_ptrs[g] + x * xs_offsets[0] + y * xs_offsets[1] +
        z * xs_offsets[2]) = quad;
  }
}

__host__
void run_expand_scattering_cross_section(
                                      const std::vector<MomentQuad*> &slgg_ptrs,
                                      const int *mat_ptr,
                                      const std::vector<MomentQuad*> &xs_ptrs,
                                      const ByteOffset slgg_offsets[2],
                                      const ByteOffset mat_offsets[3],
                                      const ByteOffset xs_offsets[3],
                                      const Rect<3> &subgrid_bounds,
                                      const int group_start)
{
  // Figure out the dimensions to launch
  const int x_range = (subgrid_bounds.hi[0] - subgrid_bounds.lo[0]) + 1; 
  const int y_range = (subgrid_bounds.hi[1] - subgrid_bounds.lo[1]) + 1;
  const int z_range = (subgrid_bounds.hi[2] - subgrid_bounds.lo[2]) + 1;

  dim3 block(gcd(x_range,32),gcd(y_range,4),gcd(z_range,4));
  dim3 grid(x_range/block.x, y_range/block.y, z_range/block.z);

  // Switch on the number of groups
  assert(slgg_ptrs.size() == xs_ptrs.size());
  // TODO: replace this template foolishness with Terra
  switch (slgg_ptrs.size())
  {
    case 1:
      {
        gpu_expand_scattering_cross_section<1><<<grid,block>>>(
                            PointerBuffer<1,MomentQuad>(slgg_ptrs), mat_ptr,
                            PointerBuffer<1,MomentQuad>(xs_ptrs), 
                            ByteOffsetArray<2>(slgg_offsets),
                            ByteOffsetArray<3>(mat_offsets),
                            ByteOffsetArray<3>(xs_offsets),
                            group_start);
        break;
      }
    case 2:
      {
        gpu_expand_scattering_cross_section<2><<<grid,block>>>(
                            PointerBuffer<2,MomentQuad>(slgg_ptrs), mat_ptr,
                            PointerBuffer<2,MomentQuad>(xs_ptrs), 
                            ByteOffsetArray<2>(slgg_offsets),
                            ByteOffsetArray<3>(mat_offsets),
                            ByteOffsetArray<3>(xs_offsets),
                            group_start);
        break;
      }
    case 3:
      {
        gpu_expand_scattering_cross_section<3><<<grid,block>>>(
                            PointerBuffer<3,MomentQuad>(slgg_ptrs), mat_ptr,
                            PointerBuffer<3,MomentQuad>(xs_ptrs), 
                            ByteOffsetArray<2>(slgg_offsets),
                            ByteOffsetArray<3>(mat_offsets),
                            ByteOffsetArray<3>(xs_offsets),
                            group_start);
        break;
      }
    case 4:
      {
        gpu_expand_scattering_cross_section<4><<<grid,block>>>(
                            PointerBuffer<4,MomentQuad>(slgg_ptrs), mat_ptr,
                            PointerBuffer<4,MomentQuad>(xs_ptrs), 
                            ByteOffsetArray<2>(slgg_offsets),
                            ByteOffsetArray<3>(mat_offsets),
                            ByteOffsetArray<3>(xs_offsets),
                            group_start);
        break;
      }
    case 5:
      {
        gpu_expand_scattering_cross_section<5><<<grid,block>>>(
                            PointerBuffer<5,MomentQuad>(slgg_ptrs), mat_ptr,
                            PointerBuffer<5,MomentQuad>(xs_ptrs), 
                            ByteOffsetArray<2>(slgg_offsets),
                            ByteOffsetArray<3>(mat_offsets),
                            ByteOffsetArray<3>(xs_offsets),
                            group_start);
        break;
      }
    case 6:
      {
        gpu_expand_scattering_cross_section<6><<<grid,block>>>(
                            PointerBuffer<6,MomentQuad>(slgg_ptrs), mat_ptr,
                            PointerBuffer<6,MomentQuad>(xs_ptrs), 
                            ByteOffsetArray<2>(slgg_offsets),
                            ByteOffsetArray<3>(mat_offsets),
                            ByteOffsetArray<3>(xs_offsets),
                            group_start);
        break;
      }
    case 7:
      {
        gpu_expand_scattering_cross_section<7><<<grid,block>>>(
                            PointerBuffer<7,MomentQuad>(slgg_ptrs), mat_ptr,
                            PointerBuffer<7,MomentQuad>(xs_ptrs), 
                            ByteOffsetArray<2>(slgg_offsets),
                            ByteOffsetArray<3>(mat_offsets),
                            ByteOffsetArray<3>(xs_offsets),
                            group_start);
        break;
      }
    case 8:
      {
        gpu_expand_scattering_cross_section<8><<<grid,block>>>(
                            PointerBuffer<8,MomentQuad>(slgg_ptrs), mat_ptr,
                            PointerBuffer<8,MomentQuad>(xs_ptrs), 
                            ByteOffsetArray<2>(slgg_offsets),
                            ByteOffsetArray<3>(mat_offsets),
                            ByteOffsetArray<3>(xs_offsets),
                            group_start);
        break;
      }
    case 9:
      {
        gpu_expand_scattering_cross_section<9><<<grid,block>>>(
                            PointerBuffer<9,MomentQuad>(slgg_ptrs), mat_ptr,
                            PointerBuffer<9,MomentQuad>(xs_ptrs), 
                            ByteOffsetArray<2>(slgg_offsets),
                            ByteOffsetArray<3>(mat_offsets),
                            ByteOffsetArray<3>(xs_offsets),
                            group_start);
        break;
      }
    case 10:
      {
        gpu_expand_scattering_cross_section<10><<<grid,block>>>(
                            PointerBuffer<10,MomentQuad>(slgg_ptrs), mat_ptr,
                            PointerBuffer<10,MomentQuad>(xs_ptrs), 
                            ByteOffsetArray<2>(slgg_offsets),
                            ByteOffsetArray<3>(mat_offsets),
                            ByteOffsetArray<3>(xs_offsets),
                            group_start);
        break;
      }
    case 11:
      {
        gpu_expand_scattering_cross_section<11><<<grid,block>>>(
                            PointerBuffer<11,MomentQuad>(slgg_ptrs), mat_ptr,
                            PointerBuffer<11,MomentQuad>(xs_ptrs), 
                            ByteOffsetArray<2>(slgg_offsets),
                            ByteOffsetArray<3>(mat_offsets),
                            ByteOffsetArray<3>(xs_offsets),
                            group_start);
        break;
      }
    case 12:
      {
        gpu_expand_scattering_cross_section<12><<<grid,block>>>(
                            PointerBuffer<12,MomentQuad>(slgg_ptrs), mat_ptr,
                            PointerBuffer<12,MomentQuad>(xs_ptrs), 
                            ByteOffsetArray<2>(slgg_offsets),
                            ByteOffsetArray<3>(mat_offsets),
                            ByteOffsetArray<3>(xs_offsets),
                            group_start);
        break;
      }
    case 13:
      {
        gpu_expand_scattering_cross_section<13><<<grid,block>>>(
                            PointerBuffer<13,MomentQuad>(slgg_ptrs), mat_ptr,
                            PointerBuffer<13,MomentQuad>(xs_ptrs), 
                            ByteOffsetArray<2>(slgg_offsets),
                            ByteOffsetArray<3>(mat_offsets),
                            ByteOffsetArray<3>(xs_offsets),
                            group_start);
        break;
      }
    case 14:
      {
        gpu_expand_scattering_cross_section<14><<<grid,block>>>(
                            PointerBuffer<14,MomentQuad>(slgg_ptrs), mat_ptr,
                            PointerBuffer<14,MomentQuad>(xs_ptrs), 
                            ByteOffsetArray<2>(slgg_offsets),
                            ByteOffsetArray<3>(mat_offsets),
                            ByteOffsetArray<3>(xs_offsets),
                            group_start);
        break;
      }
    case 15:
      {
        gpu_expand_scattering_cross_section<15><<<grid,block>>>(
                            PointerBuffer<15,MomentQuad>(slgg_ptrs), mat_ptr,
                            PointerBuffer<15,MomentQuad>(xs_ptrs), 
                            ByteOffsetArray<2>(slgg_offsets),
                            ByteOffsetArray<3>(mat_offsets),
                            ByteOffsetArray<3>(xs_offsets),
                            group_start);
        break;
      }
    case 16:
      {
        gpu_expand_scattering_cross_section<16><<<grid,block>>>(
                            PointerBuffer<16,MomentQuad>(slgg_ptrs), mat_ptr,
                            PointerBuffer<16,MomentQuad>(xs_ptrs), 
                            ByteOffsetArray<2>(slgg_offsets),
                            ByteOffsetArray<3>(mat_offsets),
                            ByteOffsetArray<3>(xs_offsets),
                            group_start);
        break;
      }
    case 24:
      {
        gpu_expand_scattering_cross_section<24><<<grid,block>>>(
                            PointerBuffer<24,MomentQuad>(slgg_ptrs), mat_ptr,
                            PointerBuffer<24,MomentQuad>(xs_ptrs), 
                            ByteOffsetArray<2>(slgg_offsets),
                            ByteOffsetArray<3>(mat_offsets),
                            ByteOffsetArray<3>(xs_offsets),
                            group_start);
        break;
      }
    case 32:
      {
        gpu_expand_scattering_cross_section<32><<<grid,block>>>(
                            PointerBuffer<32,MomentQuad>(slgg_ptrs), mat_ptr,
                            PointerBuffer<32,MomentQuad>(xs_ptrs), 
                            ByteOffsetArray<2>(slgg_offsets),
                            ByteOffsetArray<3>(mat_offsets),
                            ByteOffsetArray<3>(xs_offsets),
                            group_start);
        break;
      }
    case 40:
      {
        gpu_expand_scattering_cross_section<40><<<grid,block>>>(
                            PointerBuffer<40,MomentQuad>(slgg_ptrs), mat_ptr,
                            PointerBuffer<40,MomentQuad>(xs_ptrs), 
                            ByteOffsetArray<2>(slgg_offsets),
                            ByteOffsetArray<3>(mat_offsets),
                            ByteOffsetArray<3>(xs_offsets),
                            group_start);
        break;
      }
    case 48:
      {
        gpu_expand_scattering_cross_section<48><<<grid,block>>>(
                            PointerBuffer<48,MomentQuad>(slgg_ptrs), mat_ptr,
                            PointerBuffer<48,MomentQuad>(xs_ptrs), 
                            ByteOffsetArray<2>(slgg_offsets),
                            ByteOffsetArray<3>(mat_offsets),
                            ByteOffsetArray<3>(xs_offsets),
                            group_start);
        break;
      }
    case 56:
      {
        gpu_expand_scattering_cross_section<56><<<grid,block>>>(
                            PointerBuffer<56,MomentQuad>(slgg_ptrs), mat_ptr,
                            PointerBuffer<56,MomentQuad>(xs_ptrs), 
                            ByteOffsetArray<2>(slgg_offsets),
                            ByteOffsetArray<3>(mat_offsets),
                            ByteOffsetArray<3>(xs_offsets),
                            group_start);
        break;
      }
    case 64:
      {
        gpu_expand_scattering_cross_section<64><<<grid,block>>>(
                            PointerBuffer<64,MomentQuad>(slgg_ptrs), mat_ptr,
                            PointerBuffer<64,MomentQuad>(xs_ptrs), 
                            ByteOffsetArray<2>(slgg_offsets),
                            ByteOffsetArray<3>(mat_offsets),
                            ByteOffsetArray<3>(xs_offsets),
                            group_start);
        break;
      }
    default:
      assert(false); // add more cases
  }
}

