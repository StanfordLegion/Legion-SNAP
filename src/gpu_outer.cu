#include "hip/hip_runtime.h"
/* Copyright 2017 NVIDIA Corporation
 *
 * The U.S. Department of Energy funded the development of this software 
 * under subcontract B609478 with Lawrence Livermore National Security, LLC
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "snap_types.h"
#include "accessor.h"
#include "snap_cuda_help.h"

#include <vector>

using namespace LegionRuntime::Arrays;
using namespace LegionRuntime::Accessor;

template<int GROUPS, int STRIP_SIZE>
__global__
void gpu_flux0_outer_source(const PointerBuffer<GROUPS,double> qi0_ptrs,
                            const PointerBuffer<GROUPS,double> flux0_ptrs,
                            const PointerBuffer<GROUPS,MomentQuad> slgg_ptrs,
                            const int *mat_ptr,
                                  PointerBuffer<GROUPS,double> qo0_ptrs,
                            const ByteOffsetArray<3> qi0_offsets,
                            const ByteOffsetArray<3> flux0_offsets,
                            const ByteOffsetArray<2> slgg_offsets,
                            const ByteOffsetArray<3> mat_offsets,
                            const ByteOffsetArray<3> qo0_offsets)
{
  __shared__ double flux_buffer[GROUPS][STRIP_SIZE];
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int z = blockIdx.z;
  const int group = threadIdx.z;
  const int strip_offset = threadIdx.y * blockDim.x + threadIdx.x;
  // First, update our pointers
  const double *qi0_ptr = qi0_ptrs[group] + x * qi0_offsets[0] +
    y * qi0_offsets[1] + z * qi0_offsets[2];
  const double *flux0_ptr = flux0_ptrs[group] + x * flux0_offsets[0] +
    y * flux0_offsets[1] + z * flux0_offsets[2];
  mat_ptr += x * mat_offsets[0] + y * mat_offsets[1] + z * mat_offsets[2];
  double *qo0_ptr = qo0_ptrs[group] + x * qo0_offsets[0] + 
    y *qo0_offsets[1] + z * qo0_offsets[2];
  // Do a little prefetching of other values we need too
  // Be intelligent about loads, we're trying to keep the slgg
  // matrix in L2 cache so make sure all other loads and stores 
  // are cached with a streaming prefix
  double flux0;
  asm volatile("ld.global.cs.f64 %0, [%1];" : "=d"(flux0) : "l"(flux0_ptr) : "memory");
  // Other threads will use the material so cache at all levels
  int mat;
  asm volatile("ld.global.ca.s32 %0, [%1];" : "=r"(mat) : "l"(mat_ptr) : "memory");
  double qo0;
  asm volatile("ld.global.cs.f64 %0, [%1];" : "=d"(qo0) : "l"(qi0_ptr) : "memory");
  // Write the value into shared
  flux_buffer[group][strip_offset] = flux0;
  // Can compute our slgg_ptr with the matrix result
#ifdef LEGION_ISSUE_214_FIX
  const MomentQuad *slgg_ptr = slgg_ptrs[group] + mat * slgg_offsets[0];
#else
  const MomentQuad *slgg_ptr = slgg_ptrs[group] + (mat-1) * slgg_offsets[0];
#endif
  // Synchronize when all the writes into shared memory are done
  __syncthreads();
  // Do the math
  #pragma unroll
  for (int g = 0; g < GROUPS; g++) {
    if (g == group)
      continue;
    const MomentQuad *local_slgg = slgg_ptr + g * slgg_offsets[1];
    double cs;
    asm volatile("ld.global.ca.f64 %0, [%1];" : "=d"(cs) : "l"(local_slgg) : "memory");
    qo0 += cs * flux_buffer[g][strip_offset];
  }
  // Write out our result
  asm volatile("st.global.cs.f64 [%0], %1;" : : "l"(qo0_ptr), "d"(qo0) : "memory");
}

template<int GROUPS, int MAX_X, int MAX_Y>
__host__
void flux0_launch_helper(Rect<3> subgrid_bounds,
                         const std::vector<double*> &qi0_ptrs,
                         const std::vector<double*> &flux0_ptrs,
                         const std::vector<MomentQuad*> &slgg_ptrs,
                         const std::vector<double*> &qo0_ptrs, 
                         const int *mat_ptr, 
                         const ByteOffset qi0_offsets[3], 
                         const ByteOffset flux0_offsets[3],
                         const ByteOffset slgg_offsets[2], 
                         const ByteOffset qo0_offsets[3],
                         const ByteOffset mat_offsets[3])
{
  const int x_range = (subgrid_bounds.hi[0] - subgrid_bounds.lo[0]) + 1;
  const int y_range = (subgrid_bounds.hi[1] - subgrid_bounds.lo[1]) + 1;
  const int z_range = (subgrid_bounds.hi[2] - subgrid_bounds.lo[2]) + 1;
  dim3 block(gcd(x_range,MAX_X), gcd(y_range,MAX_Y), GROUPS);
  dim3 grid(x_range/block.x, y_range/block.y, z_range);
  gpu_flux0_outer_source<GROUPS,MAX_X*MAX_Y><<<grid,block>>>(
                              PointerBuffer<GROUPS,double>(qi0_ptrs),
                              PointerBuffer<GROUPS,double>(flux0_ptrs),
                              PointerBuffer<GROUPS,MomentQuad>(slgg_ptrs), mat_ptr,
                              PointerBuffer<GROUPS,double>(qo0_ptrs),
                              ByteOffsetArray<3>(qi0_offsets),
                              ByteOffsetArray<3>(flux0_offsets),
                              ByteOffsetArray<2>(slgg_offsets),
                              ByteOffsetArray<3>(mat_offsets),
                              ByteOffsetArray<3>(qo0_offsets));
}

__host__
void run_flux0_outer_source(Rect<3> subgrid_bounds,
                            const std::vector<double*> &qi0_ptrs,
                            const std::vector<double*> &flux0_ptrs,
                            const std::vector<MomentQuad*> &slgg_ptrs,
                            const std::vector<double*> &qo0_ptrs, 
                            const int *mat_ptr, 
                            const ByteOffset qi0_offsets[3], 
                            const ByteOffset flux0_offsets[3],
                            const ByteOffset slgg_offsets[2], 
                            const ByteOffset qo0_offsets[3],
                            const ByteOffset mat_offsets[3], const int num_groups)
{
  // TODO: replace this template madness with Terra
#define GROUP_CASE(g,x,y)                                                           \
  case g:                                                                           \
    {                                                                               \
      flux0_launch_helper<g,x,y>(subgrid_bounds, qi0_ptrs, flux0_ptrs, slgg_ptrs,   \
                               qo0_ptrs, mat_ptr, qi0_offsets, flux0_offsets,       \
                               slgg_offsets, qo0_offsets, mat_offsets);             \
      break;                                                                        \
    }
  switch (num_groups)
  {
    GROUP_CASE(1,32,32)
    GROUP_CASE(2,32,16)
    GROUP_CASE(3,32,8)
    GROUP_CASE(4,32,8)
    GROUP_CASE(5,32,4)
    GROUP_CASE(6,32,4)
    GROUP_CASE(7,32,4)
    GROUP_CASE(8,32,4)
    GROUP_CASE(9,32,2)
    GROUP_CASE(10,32,2)
    GROUP_CASE(11,32,2)
    GROUP_CASE(12,32,2)
    GROUP_CASE(13,32,2)
    GROUP_CASE(14,32,2)
    GROUP_CASE(15,32,2)
    GROUP_CASE(16,32,2)
    GROUP_CASE(17,16,2)
    GROUP_CASE(18,16,2)
    GROUP_CASE(19,16,2)
    GROUP_CASE(20,16,2)
    GROUP_CASE(21,16,2)
    GROUP_CASE(22,16,2)
    GROUP_CASE(23,16,2)
    GROUP_CASE(24,16,2)
    GROUP_CASE(25,16,2)
    GROUP_CASE(26,16,2)
    GROUP_CASE(27,16,2)
    GROUP_CASE(28,16,2)
    GROUP_CASE(29,16,2)
    GROUP_CASE(30,16,2)
    GROUP_CASE(31,16,2)
    GROUP_CASE(32,16,2)
    GROUP_CASE(33,16,1)
    GROUP_CASE(34,16,1)
    GROUP_CASE(35,16,1)
    GROUP_CASE(36,16,1)
    GROUP_CASE(37,16,1)
    GROUP_CASE(38,16,1)
    GROUP_CASE(39,16,1)
    GROUP_CASE(40,16,1)
    GROUP_CASE(41,16,1)
    GROUP_CASE(42,16,1)
    GROUP_CASE(43,16,1)
    GROUP_CASE(44,16,1)
    GROUP_CASE(45,16,1)
    GROUP_CASE(46,16,1)
    GROUP_CASE(47,16,1)
    GROUP_CASE(48,16,1)
    GROUP_CASE(49,16,1)
    GROUP_CASE(50,16,1)
    GROUP_CASE(51,16,1)
    GROUP_CASE(52,16,1)
    GROUP_CASE(53,16,1)
    GROUP_CASE(54,16,1)
    GROUP_CASE(55,16,1)
    GROUP_CASE(56,16,1)
    GROUP_CASE(57,16,1)
    GROUP_CASE(58,16,1)
    GROUP_CASE(59,16,1)
    GROUP_CASE(60,16,1)
    GROUP_CASE(61,16,1)
    GROUP_CASE(62,16,1)
    GROUP_CASE(63,16,1)
    GROUP_CASE(64,16,1)
    GROUP_CASE(65,8,1)
    GROUP_CASE(66,8,1)
    GROUP_CASE(67,8,1)
    GROUP_CASE(68,8,1)
    GROUP_CASE(69,8,1)
    GROUP_CASE(70,8,1)
    GROUP_CASE(71,8,1)
    GROUP_CASE(72,8,1)
    GROUP_CASE(73,8,1)
    GROUP_CASE(74,8,1)
    GROUP_CASE(75,8,1)
    GROUP_CASE(76,8,1)
    GROUP_CASE(77,8,1)
    GROUP_CASE(78,8,1)
    GROUP_CASE(79,8,1)
    GROUP_CASE(80,8,1)
    GROUP_CASE(81,8,1)
    GROUP_CASE(82,8,1)
    GROUP_CASE(83,8,1)
    GROUP_CASE(84,8,1)
    GROUP_CASE(85,8,1)
    GROUP_CASE(86,8,1)
    GROUP_CASE(87,8,1)
    GROUP_CASE(88,8,1)
    GROUP_CASE(89,8,1)
    GROUP_CASE(90,8,1)
    GROUP_CASE(91,8,1)
    GROUP_CASE(92,8,1)
    GROUP_CASE(93,8,1)
    GROUP_CASE(94,8,1)
    GROUP_CASE(95,8,1)
    GROUP_CASE(96,8,1)
    // About to drop down to 1 CTA per SM due to shared memory
    default:
      printf("Adding group case to outer flux0 computation!\n");
      assert(false);
  }
#undef GROUP_CASE
}

template<int GROUPS, int STRIP_SIZE>
__global__
void gpu_fluxm_outer_source(const PointerBuffer<GROUPS,MomentTriple> fluxm_ptrs,
                            const PointerBuffer<GROUPS,MomentQuad> slgg_ptrs,
                            const int           *mat_ptr,
                                  PointerBuffer<GROUPS,MomentTriple> qom_ptrs,
                            ByteOffsetArray<3> fluxm_offsets,
                            ByteOffsetArray<2> slgg_offsets,
                            ByteOffsetArray<3> mat_offsets,
                            ByteOffsetArray<3> qom_offsets,
                            const int           num_moments,
                            const ConstBuffer<4,int> lma)
{
  __shared__ double fluxm_buffer_0[GROUPS][STRIP_SIZE];
  __shared__ double fluxm_buffer_1[GROUPS][STRIP_SIZE];
  __shared__ double fluxm_buffer_2[GROUPS][STRIP_SIZE];
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y + blockDim.y + threadIdx.y;
  const int z = blockIdx.z;
  const int group = threadIdx.z;
  const int strip_offset = threadIdx.y * blockDim.x + threadIdx.x;
  const MomentTriple *fluxm_ptr = fluxm_ptrs[group] + x * fluxm_offsets[0] +
    y * fluxm_offsets[1] + z * fluxm_offsets[2];
  mat_ptr += x * mat_offsets[0] + y * mat_offsets[1] + z * mat_offsets[2];
  MomentTriple *qom_ptr = qom_ptrs[group] + x * qom_offsets[0] + 
    y *qom_offsets[1] + z * qom_offsets[2];
  MomentTriple fluxm;
  asm volatile("ld.global.cs.v2.f64 {%0,%1}, [%2];" : "=d"(fluxm[0]), "=d"(fluxm[1]) 
                : "l"(fluxm_ptr) : "memory");
  asm volatile("ld.global.cs.f64 %0, [%1];" : "=d"(fluxm[2]) 
                : "l"(((char*)fluxm_ptr)+16) : "memory");
  int mat;
  asm volatile("ld.global.ca.s32 %0, [%1];" : "=r"(mat) : "l"(mat_ptr) : "memory");
  // Write the fluxm into shared memory
  fluxm_buffer_0[group][strip_offset] = fluxm[0];
  fluxm_buffer_1[group][strip_offset] = fluxm[1];
  fluxm_buffer_2[group][strip_offset] = fluxm[2];
  // Can compute our slgg_ptr with the matrix result
  const MomentQuad *slgg_ptr = slgg_ptrs[group] + mat * slgg_offsets[0];
  // Synchronize to make sure all the writes to shared are done 
  __syncthreads();
  // Do the math
  MomentTriple qom;
  #pragma unroll
  for (int g = 0; g < GROUPS; g++) {
    if (g == group)
      continue;
    int moment = 0;
    const MomentQuad *local_slgg = slgg_ptr + g * slgg_offsets[1];
    MomentQuad scat;
    asm volatile("ld.global.ca.v2.f64 {%0,%1}, [%2];" : "=d"(scat[0]), "=d"(scat[1])
                  : "l"(local_slgg) : "memory");
    asm volatile("ld.global.ca.v2.f64 {%0,%1}, [%2];" : "=d"(scat[2]), "=d"(scat[3])
                  : "l"(((char*)local_slgg)+16) : "memory");
    MomentTriple csm;
    for (int l = 1; l < num_moments; l++) {
      for (int j = 0; j < lma[l]; j++)
        csm[moment+j] = scat[l];
      moment += lma[l];
    }
    fluxm[0] = fluxm_buffer_0[g][strip_offset];
    fluxm[1] = fluxm_buffer_1[g][strip_offset];
    fluxm[2] = fluxm_buffer_2[g][strip_offset];
    for (int l = 0; l < (num_moments-1); l++)
      qom[l] += csm[l] * fluxm[l];
  }
  // Now we can write out the result
  asm volatile("st.global.cs.v2.f64 [%0], {%1,%2};" : : "l"(qom_ptr), 
                "d"(qom[0]), "d"(qom[1]) : "memory");
  asm volatile("st.global.cs.f64 [%0], %1;" : : "l"(((char*)qom_ptr)+16),
                "d"(qom[2]) : "memory");
}

template<int GROUPS, int MAX_X, int MAX_Y>
__host__
void fluxm_launch_helper(Rect<3> subgrid_bounds,
                         const std::vector<MomentTriple*> &fluxm_ptrs,
                         const std::vector<MomentQuad*> &slgg_ptrs,
                         const std::vector<MomentTriple*> &qom_ptrs, 
                         const int *mat_ptr, 
                         const ByteOffset fluxm_offsets[3], 
                         const ByteOffset slgg_offsets[2],
                         const ByteOffset mat_offsets[3], 
                         const ByteOffset qom_offsets[3],
                         const int num_groups, const int num_moments, const int lma[4])
{
  const int x_range = (subgrid_bounds.hi[0] - subgrid_bounds.lo[0]) + 1;
  const int y_range = (subgrid_bounds.hi[1] - subgrid_bounds.lo[1]) + 1;
  const int z_range = (subgrid_bounds.hi[2] - subgrid_bounds.lo[2]) + 1;
  dim3 block(gcd(x_range,MAX_X), gcd(y_range,MAX_Y), GROUPS);
  dim3 grid(x_range/block.x, y_range/block.y, z_range);
  gpu_fluxm_outer_source<GROUPS,MAX_X*MAX_Y><<<grid,block>>>(
                            PointerBuffer<GROUPS,MomentTriple>(fluxm_ptrs), 
                            PointerBuffer<GROUPS,MomentQuad>(slgg_ptrs), mat_ptr,
                            PointerBuffer<GROUPS,MomentTriple>(qom_ptrs),
                            ByteOffsetArray<3>(fluxm_offsets),
                            ByteOffsetArray<2>(slgg_offsets),
                            ByteOffsetArray<3>(mat_offsets),
                            ByteOffsetArray<3>(qom_offsets),
                            num_moments, ConstBuffer<4,int>(lma));
}

__host__
void run_fluxm_outer_source(Rect<3> subgrid_bounds,
                            const std::vector<MomentTriple*> &fluxm_ptrs,
                            const std::vector<MomentQuad*> &slgg_ptrs,
                            const std::vector<MomentTriple*> &qom_ptrs, 
                            const int *mat_ptr, 
                            const ByteOffset fluxm_offsets[3], 
                            const ByteOffset slgg_offsets[2],
                            const ByteOffset mat_offsets[3], 
                            const ByteOffset qom_offsets[3],
                            const int num_groups, const int num_moments, const int lma[4])
{
  // TODO: replace this template madness with Terra
#define GROUP_CASE(g,x,y)                                                         \
  case g:                                                                         \
    {                                                                             \
      fluxm_launch_helper<g,x,y>(subgrid_bounds, fluxm_ptrs, slgg_ptrs, qom_ptrs, \
                             mat_ptr, fluxm_offsets, slgg_offsets, mat_offsets,   \
                             qom_offsets, num_groups, num_moments, lma);          \
      break;                                                                      \
    }
  switch (num_groups)
  {
    GROUP_CASE(1,32,32)
    GROUP_CASE(2,32,16)
    GROUP_CASE(3,32,8)
    GROUP_CASE(4,32,8)
    GROUP_CASE(5,32,4)
    GROUP_CASE(6,32,4)
    GROUP_CASE(7,32,4)
    GROUP_CASE(8,32,4)
    GROUP_CASE(9,32,2)
    GROUP_CASE(10,32,2)
    GROUP_CASE(11,32,2)
    GROUP_CASE(12,32,2)
    GROUP_CASE(13,32,2)
    GROUP_CASE(14,32,2)
    GROUP_CASE(15,32,2)
    GROUP_CASE(16,32,2)
    GROUP_CASE(17,16,2)
    GROUP_CASE(18,16,2)
    GROUP_CASE(19,16,2)
    GROUP_CASE(20,16,2)
    GROUP_CASE(21,16,2)
    GROUP_CASE(22,16,2)
    GROUP_CASE(23,16,2)
    GROUP_CASE(24,16,2)
    GROUP_CASE(25,16,2)
    GROUP_CASE(26,16,2)
    GROUP_CASE(27,16,2)
    GROUP_CASE(28,16,2)
    GROUP_CASE(29,16,2)
    GROUP_CASE(30,16,2)
    GROUP_CASE(31,16,2)
    GROUP_CASE(32,16,2)
    GROUP_CASE(33,16,1)
    GROUP_CASE(34,16,1)
    GROUP_CASE(35,16,1)
    GROUP_CASE(36,16,1)
    GROUP_CASE(37,16,1)
    GROUP_CASE(38,16,1)
    GROUP_CASE(39,16,1)
    GROUP_CASE(40,16,1)
    GROUP_CASE(41,16,1)
    GROUP_CASE(42,16,1)
    GROUP_CASE(43,16,1)
    GROUP_CASE(44,16,1)
    GROUP_CASE(45,16,1)
    GROUP_CASE(46,16,1)
    GROUP_CASE(47,16,1)
    GROUP_CASE(48,16,1)
    GROUP_CASE(49,16,1)
    GROUP_CASE(50,16,1)
    GROUP_CASE(51,16,1)
    GROUP_CASE(52,16,1)
    GROUP_CASE(53,16,1)
    GROUP_CASE(54,16,1)
    GROUP_CASE(55,16,1)
    GROUP_CASE(56,16,1)
    GROUP_CASE(57,16,1)
    GROUP_CASE(58,16,1)
    GROUP_CASE(59,16,1)
    GROUP_CASE(60,16,1)
    GROUP_CASE(61,16,1)
    GROUP_CASE(62,16,1)
    GROUP_CASE(63,16,1)
    GROUP_CASE(64,16,1)
    GROUP_CASE(65,8,1)
    GROUP_CASE(66,8,1)
    GROUP_CASE(67,8,1)
    GROUP_CASE(68,8,1)
    GROUP_CASE(69,8,1)
    GROUP_CASE(70,8,1)
    GROUP_CASE(71,8,1)
    GROUP_CASE(72,8,1)
    GROUP_CASE(73,8,1)
    GROUP_CASE(74,8,1)
    GROUP_CASE(75,8,1)
    GROUP_CASE(76,8,1)
    GROUP_CASE(77,8,1)
    GROUP_CASE(78,8,1)
    GROUP_CASE(79,8,1)
    GROUP_CASE(80,8,1)
    GROUP_CASE(81,8,1)
    GROUP_CASE(82,8,1)
    GROUP_CASE(83,8,1)
    GROUP_CASE(84,8,1)
    GROUP_CASE(85,8,1)
    GROUP_CASE(86,8,1)
    GROUP_CASE(87,8,1)
    GROUP_CASE(88,8,1)
    GROUP_CASE(89,8,1)
    GROUP_CASE(90,8,1)
    GROUP_CASE(91,8,1)
    GROUP_CASE(92,8,1)
    GROUP_CASE(93,8,1)
    GROUP_CASE(94,8,1)
    GROUP_CASE(95,8,1)
    GROUP_CASE(96,8,1)
    default:
      printf("Adding group case to outer fluxm computation!\n");
      assert(false);
  }
#undef GROUP_CASE
}

__global__
void gpu_outer_convergence(const double *flux0_ptr, const double *flux0po_ptr,
                           ByteOffsetArray<3> flux0_offsets,
                           ByteOffsetArray<3> flux0po_offsets,
                           const double epsi, int *total_converged)
{
  // We know there is never more than 32 warps in a CTA
  __shared__ int trampoline[32];

  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int z = blockIdx.z * blockDim.z + threadIdx.z;

  flux0_ptr += x * flux0_offsets[0] + y * flux0_offsets[1] + z * flux0_offsets[2];
  flux0po_ptr += x * flux0po_offsets[0] + y * flux0po_offsets[1] + z * flux0po_offsets[2];

  const double tolr = 1.0e-12;
  
  double flux0po = *flux0po_ptr;
  double df = 1.0;
  if (fabs(flux0po) < tolr) {
    flux0po = 1.0;
    df = 0.0;
  }
  double flux0 = *flux0_ptr;
  df = fabs( (flux0 / flux0po) - df );
  int local_converged = 1;
  if ((df >= -INFINITY) && (df > epsi))
    local_converged = 0;
  // Perform a local reduction inside the CTA
  // Butterfly reduction across all threads in all warps
  unsigned laneid;
  asm volatile("mov.u32 %0, %laneid;" : "=r"(laneid) : );
  const unsigned warpid = 
    ((threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x) >> 5;
  for (int i = 16; i >= 1; i/=2)
    local_converged += __shfl_xor(local_converged, i, 32);
  // Initialize the trampoline
  if (warpid == 0)
    trampoline[laneid] = 0;
  __syncthreads();
  // First thread in each warp writes out all values
  if (laneid == 0)
    trampoline[warpid] = local_converged;
  __syncthreads();
  // Butterfly reduction across all thread in the first warp
  if (warpid == 0) {
    local_converged = trampoline[laneid];
    for (int i = 16; i >= 1; i/=2)
      local_converged += __shfl_xor(local_converged, i, 32);
    // First thread does the atomic
    if (laneid == 0)
      atomicAdd(total_converged, local_converged);
  }
}

__host__
bool run_outer_convergence(Rect<3> subgrid_bounds,
                           const std::vector<double*> flux0_ptrs,
                           const std::vector<double*> flux0po_ptrs,
                           const ByteOffset flux0_offsets[3], 
                           const ByteOffset flux0po_offsets[3],
                           const double epsi)
{
  int *converged_d;
  hipMalloc((void**)&converged_d, sizeof(int));
  // Initialize the result
  hipMemset(converged_d, 0/*value*/, 1/*count*/); 
  // Launch the kernels
  const int x_range = (subgrid_bounds.hi[0] - subgrid_bounds.lo[0]) + 1;
  const int y_range = (subgrid_bounds.hi[1] - subgrid_bounds.lo[1]) + 1;
  const int z_range = (subgrid_bounds.hi[2] - subgrid_bounds.lo[2]) + 1;
  dim3 block(gcd(x_range,32),gcd(y_range,4),gcd(z_range,4));
  dim3 grid(x_range/block.x, y_range/block.y, z_range/block.z);

  assert(flux0_ptrs.size() == flux0po_ptrs.size());
  for (unsigned idx = 0; idx < flux0_ptrs.size(); idx++) {
    gpu_outer_convergence<<<grid,block>>>(flux0_ptrs[idx], flux0po_ptrs[idx],
                                          ByteOffsetArray<3>(flux0_offsets),
                                          ByteOffsetArray<3>(flux0po_offsets),
                                          epsi, converged_d); 
  }
  // Copy back: CUDA hijack synchronizes for us
  int converged_h;
  hipMemcpy(&converged_h, converged_d, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(converged_d);
  // We've converged if the total converged points are the number of tests
  return (converged_h == int(x_range * y_range * z_range * flux0_ptrs.size()));
}

