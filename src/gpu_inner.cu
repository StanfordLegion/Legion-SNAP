#include "hip/hip_runtime.h"
/* Copyright 2017 NVIDIA Corporation
 *
 * The U.S. Department of Energy funded the development of this software 
 * under subcontract B609478 with Lawrence Livermore National Security, LLC
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "snap_types.h"
#include "accessor.h"
#include "snap_cuda_help.h"

using namespace LegionRuntime::Arrays;
using namespace LegionRuntime::Accessor;

__global__
void gpu_inner_source_single_moment(const MomentQuad  *sxs_ptr,
                                    const double      *flux0_ptr,
                                    const double      *q2grp0_ptr,
                                          MomentQuad  *qtot_ptr,
                                    ByteOffsetArray<3> sxs_offsets,
                                    ByteOffsetArray<3> flux0_offsets,
                                    ByteOffsetArray<3> q2grp0_offsets,
                                    ByteOffsetArray<3> qtot_offsets)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int z = blockIdx.z * blockDim.z + threadIdx.z;
  // Straight up data parallel so nothing interesting to do
  sxs_ptr += x * sxs_offsets[0] + y * sxs_offsets[1] + z * sxs_offsets[2];
  MomentQuad sxs_quad = *sxs_ptr;

  flux0_ptr += x * flux0_offsets[0] + y * flux0_offsets[1] + z * flux0_offsets[2];
  double flux0 = *flux0_ptr;

  q2grp0_ptr += x * q2grp0_offsets[0] + y * q2grp0_offsets[1] + z * q2grp0_offsets[2];
  double q0 = *q2grp0_ptr;

  MomentQuad quad;
  quad[0] = q0 + flux0 * sxs_quad[0]; 

  qtot_ptr += x * qtot_offsets[0] + y * qtot_offsets[1] + z * qtot_offsets[2];
  *qtot_ptr = quad;
}

__host__
void run_inner_source_single_moment(Rect<3>           subgrid_bounds,
                                    const MomentQuad  *sxs_ptr,
                                    const double      *flux0_ptr,
                                    const double      *q2grp0_ptr,
                                          MomentQuad  *qtot_ptr,
                                    const ByteOffset  sxs_offsets[3],
                                    const ByteOffset  flux0_offsets[3],
                                    const ByteOffset  q2grp0_offsets[3],
                                    const ByteOffset  qtot_offsets[3])
{
  const int x_range = (subgrid_bounds.hi[0] - subgrid_bounds.lo[0]) + 1;
  const int y_range = (subgrid_bounds.hi[1] - subgrid_bounds.lo[1]) + 1;
  const int z_range = (subgrid_bounds.hi[2] - subgrid_bounds.lo[2]) + 1;

  dim3 block(gcd(x_range,32),gcd(y_range,4),gcd(z_range,4));
  dim3 grid(x_range/block.x, y_range/block.y, z_range/block.z);
  gpu_inner_source_single_moment<<<grid,block>>>(sxs_ptr, flux0_ptr,
                                                 q2grp0_ptr, qtot_ptr, 
                                                 ByteOffsetArray<3>(sxs_offsets),
                                                 ByteOffsetArray<3>(flux0_offsets),
                                                 ByteOffsetArray<3>(q2grp0_offsets),
                                                 ByteOffsetArray<3>(qtot_offsets));
}

__global__
void gpu_inner_source_multi_moment(const MomentQuad   *sxs_ptr,
                                   const double       *flux0_ptr,
                                   const double       *q2grp0_ptr,
                                   const MomentTriple *fluxm_ptr,
                                   const MomentTriple *q2grpm_ptr,
                                         MomentQuad   *qtot_ptr,
                                   ByteOffsetArray<3> sxs_offsets,
                                   ByteOffsetArray<3> flux0_offsets,
                                   ByteOffsetArray<3> q2grp0_offsets,
                                   ByteOffsetArray<3> fluxm_offsets,
                                   ByteOffsetArray<3> q2grpm_offsets,
                                   ByteOffsetArray<3> qtot_offsets,
                                   const int num_moments, 
                                   const ConstBuffer<4,int> lma)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int z = blockIdx.z * blockDim.z + threadIdx.z;
  // Straight up data parallel so nothing interesting to do
  sxs_ptr += x * sxs_offsets[0] + y * sxs_offsets[1] + z * sxs_offsets[2];
  MomentQuad sxs_quad = *sxs_ptr;

  flux0_ptr += x * flux0_offsets[0] + y * flux0_offsets[1] + z * flux0_offsets[2];
  double flux0 = *flux0_ptr;

  q2grp0_ptr += x * q2grp0_offsets[0] + y * q2grp0_offsets[1] + z * q2grp0_offsets[2];
  double q0 = *q2grp0_ptr;

  fluxm_ptr += x * fluxm_offsets[0] + y * fluxm_offsets[1] + z * fluxm_offsets[2];
  MomentTriple fluxm = *fluxm_ptr;

  q2grpm_ptr += x * q2grpm_offsets[0] + y * q2grpm_offsets[1] + z * q2grpm_offsets[2];
  MomentTriple qom = *q2grpm_ptr;

  MomentQuad quad;
  quad[0] = q0 + flux0 * sxs_quad[0]; 
  
  int moment = 0;
  for (int l = 1; l < num_moments; l++) {
    for (int i = 0; i < lma[l]; i++)
      quad[moment+i+1] = qom[moment+i] + fluxm[moment+i] * sxs_quad[l];
    moment += lma[l];
  }

  qtot_ptr += x * qtot_offsets[0] + y * qtot_offsets[1] + z * qtot_offsets[2];
  *qtot_ptr = quad;
}

__host__
void run_inner_source_multi_moment(Rect<3> subgrid_bounds,
                                   const MomentQuad   *sxs_ptr,
                                   const double       *flux0_ptr,
                                   const double       *q2grp0_ptr,
                                   const MomentTriple *fluxm_ptr,
                                   const MomentTriple *q2grpm_ptr,
                                         MomentQuad   *qtot_ptr,
                                   const ByteOffset   sxs_offsets[3],
                                   const ByteOffset   flux0_offsets[3],
                                   const ByteOffset   q2grp0_offsets[3],
                                   const ByteOffset   fluxm_offsets[3],
                                   const ByteOffset   q2grpm_offsets[3],
                                   const ByteOffset   qtot_offsets[3],
                                   const int num_moments, const int lma[4])
{
  const int x_range = (subgrid_bounds.hi[0] - subgrid_bounds.lo[0]) + 1;
  const int y_range = (subgrid_bounds.hi[1] - subgrid_bounds.lo[1]) + 1;
  const int z_range = (subgrid_bounds.hi[2] - subgrid_bounds.lo[2]) + 1;

  dim3 block(gcd(x_range,32),gcd(y_range,4),gcd(z_range,4));
  dim3 grid(x_range/block.x, y_range/block.y, z_range/block.z);
  gpu_inner_source_multi_moment<<<grid,block>>>(sxs_ptr, flux0_ptr, q2grp0_ptr, 
                                                fluxm_ptr, q2grpm_ptr, qtot_ptr, 
                                                ByteOffsetArray<3>(sxs_offsets),
                                                ByteOffsetArray<3>(flux0_offsets),
                                                ByteOffsetArray<3>(q2grp0_offsets),
                                                ByteOffsetArray<3>(fluxm_offsets),
                                                ByteOffsetArray<3>(q2grpm_offsets),
                                                ByteOffsetArray<3>(qtot_offsets),
                                                num_moments, ConstBuffer<4,int>(lma));
}

__global__
void gpu_inner_convergence(const double *flux0_ptr, const double *flux0pi_ptr,
                           ByteOffsetArray<3> flux0_offsets,
                           ByteOffsetArray<3> flux0pi_offsets,
                           const double epsi, int *total_converged)
{
  // We know there is never more than 32 warps in a CTA
  __shared__ int trampoline[32];

  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int z = blockIdx.z * blockDim.z + threadIdx.z;

  flux0_ptr += x * flux0_offsets[0] + y * flux0_offsets[1] + z * flux0_offsets[2];
  flux0pi_ptr += x * flux0pi_offsets[0] + y * flux0pi_offsets[1] + z * flux0pi_offsets[2];

  const double tolr = 1.0e-12;

  double flux0pi = *flux0pi_ptr;
  double df = 1.0;
  if (fabs(flux0pi) < tolr) {
    flux0pi = 1.0;
    df = 0.0;
  }
  double flux0 = *flux0_ptr;
  df = fabs( (flux0 / flux0pi) - df );
  int local_converged = 1;
  if ((df >= -INFINITY) && (df > epsi))
    local_converged = 0;
  // Perform a local reduction inside the CTA
  // Butterfly reduction across all threads in all warps
  unsigned laneid;
  asm volatile("mov.u32 %0, %laneid;" : "=r"(laneid) : );
  const unsigned warpid = 
    ((threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x) >> 5;
  for (int i = 16; i >= 1; i/=2)
    local_converged += __shfl_xor(local_converged, i, 32);
  // Initialize the trampoline
  if (warpid == 0)
    trampoline[laneid] = 0;
  __syncthreads();
  // First thread in each warp writes out all values
  if (laneid == 0)
    trampoline[warpid] = local_converged;
  __syncthreads();
  // Butterfly reduction across all thread in the first warp
  if (warpid == 0) {
    local_converged = trampoline[laneid];
    for (int i = 16; i >= 1; i/=2)
      local_converged += __shfl_xor(local_converged, i, 32);
    // First thread does the atomic
    if (laneid == 0)
      atomicAdd(total_converged, local_converged);
  }
}

__host__
bool run_inner_convergence(Rect<3> subgrid_bounds,
                           const std::vector<double*> flux0_ptrs,
                           const std::vector<double*> flux0pi_ptrs,
                           const ByteOffset flux0_offsets[3], 
                           const ByteOffset flux0pi_offsets[3],
                           const double epsi)
{
  int *converged_d;
  hipMalloc((void**)&converged_d, sizeof(int));
  // Initialize the result
  hipMemset(converged_d, 0/*value*/, 1/*count*/); 
  // Launch the kernels
  const int x_range = (subgrid_bounds.hi[0] - subgrid_bounds.lo[0]) + 1;
  const int y_range = (subgrid_bounds.hi[1] - subgrid_bounds.lo[1]) + 1;
  const int z_range = (subgrid_bounds.hi[2] - subgrid_bounds.lo[2]) + 1;

  dim3 block(gcd(x_range,32),gcd(y_range,4),gcd(z_range,4));
  dim3 grid(x_range/block.x, y_range/block.y, z_range/block.z);

  assert(flux0_ptrs.size() == flux0pi_ptrs.size());
  for (unsigned idx = 0; idx < flux0_ptrs.size(); idx++) {
    gpu_inner_convergence<<<grid,block>>>(flux0_ptrs[idx], flux0pi_ptrs[idx],
                                          ByteOffsetArray<3>(flux0_offsets),
                                          ByteOffsetArray<3>(flux0pi_offsets),
                                          epsi, converged_d); 
  }
  // Copy back: CUDA hijack synchronizes for us
  int converged_h;
  hipMemcpy(&converged_h, converged_d, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(converged_d);
  // We've converged if the total converged points are the number of tests
  return (converged_h == int(x_range * y_range * z_range * flux0_ptrs.size()));
}

