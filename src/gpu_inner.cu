#include "hip/hip_runtime.h"
/* Copyright 2017 NVIDIA Corporation
 *
 * The U.S. Department of Energy funded the development of this software 
 * under subcontract B609478 with Lawrence Livermore National Security, LLC
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "snap.h"
#include "snap_cuda_help.h"

__global__
void gpu_inner_source_single_moment(const Point<3> origin,
                                    const Accessor<MomentQuad,3> fa_sxs,
                                    const Accessor<double,3> fa_flux0,
                                    const Accessor<double,3> fa_q2grp0,
                                          Accessor<MomentQuad,3> fa_qtot)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int z = blockIdx.z * blockDim.z + threadIdx.z;
  const Point<3> p = origin + Point<3>(x,y,z);

  // Straight up data parallel so nothing interesting to do
  MomentQuad sxs_quad = fa_sxs[p];
  double flux0 = fa_flux0[p];
  double q0 = fa_q2grp0[p];

  MomentQuad quad;
  quad[0] = q0 + flux0 * sxs_quad[0]; 
  fa_qtot[p] = quad;
}

__host__
void run_inner_source_single_moment(const Rect<3> subgrid_bounds,
                                    const Accessor<MomentQuad,3> fa_sxs,
                                    const Accessor<double,3> fa_flux0,
                                    const Accessor<double,3> fa_q2grp0,
                                          Accessor<MomentQuad,3> fa_qtot)
{
  const int x_range = (subgrid_bounds.hi[0] - subgrid_bounds.lo[0]) + 1;
  const int y_range = (subgrid_bounds.hi[1] - subgrid_bounds.lo[1]) + 1;
  const int z_range = (subgrid_bounds.hi[2] - subgrid_bounds.lo[2]) + 1;

  dim3 block(gcd(x_range,32),gcd(y_range,4),gcd(z_range,4));
  dim3 grid(x_range/block.x, y_range/block.y, z_range/block.z);
  gpu_inner_source_single_moment<<<grid,block>>>(subgrid_bounds.lo,
                                                 fa_sxs, fa_flux0, 
                                                 fa_q2grp0, fa_qtot);
}

__global__
void gpu_inner_source_multi_moment(const Point<3> origin,
                                   const Accessor<MomentQuad,3> fa_sxs,
                                   const Accessor<double,3> fa_flux0,
                                   const Accessor<double,3> fa_q2grp0,
                                   const Accessor<MomentTriple,3> fa_fluxm,
                                   const Accessor<MomentTriple,3> fa_q2grpm,
                                         Accessor<MomentQuad,3> fa_qtot,
                                   const int num_moments,
                                   const ConstBuffer<4,int> lma)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int z = blockIdx.z * blockDim.z + threadIdx.z;
  const Point<3> p = origin + Point<3>(x,y,z);
  // Straight up data parallel so nothing interesting to do 
  MomentQuad sxs_quad = fa_sxs[p];
  double flux0 = fa_flux0[p];
  double q0 = fa_q2grp0[p];
  MomentTriple fluxm = fa_fluxm[p];
  MomentTriple qom = fa_q2grpm[p];

  MomentQuad quad;
  quad[0] = q0 + flux0 * sxs_quad[0]; 
  
  int moment = 0;
  for (int l = 1; l < num_moments; l++) {
    for (int i = 0; i < lma[l]; i++)
      quad[moment+i+1] = qom[moment+i] + fluxm[moment+i] * sxs_quad[l];
    moment += lma[l];
  }

  fa_qtot[p] = quad;
}

__host__
void run_inner_source_multi_moment(const Rect<3> subgrid_bounds,
                                   const Accessor<MomentQuad,3> fa_sxs,
                                   const Accessor<double,3> fa_flux0,
                                   const Accessor<double,3> fa_q2grp0,
                                   const Accessor<MomentTriple,3> fa_fluxm,
                                   const Accessor<MomentTriple,3> fa_q2grpm,
                                         Accessor<MomentQuad,3> fa_qtot,
                                   const int num_moments, const int lma[4])
{
  const int x_range = (subgrid_bounds.hi[0] - subgrid_bounds.lo[0]) + 1;
  const int y_range = (subgrid_bounds.hi[1] - subgrid_bounds.lo[1]) + 1;
  const int z_range = (subgrid_bounds.hi[2] - subgrid_bounds.lo[2]) + 1;

  dim3 block(gcd(x_range,32),gcd(y_range,4),gcd(z_range,4));
  dim3 grid(x_range/block.x, y_range/block.y, z_range/block.z);
  gpu_inner_source_multi_moment<<<grid,block>>>(subgrid_bounds.lo,
                                                fa_sxs, fa_flux0, fa_q2grp0,
                                                fa_fluxm, fa_q2grpm, fa_qtot,
                                                num_moments, ConstBuffer<4,int>(lma));
}

__global__
void gpu_inner_convergence(const Point<3> origin,
                           const Accessor<double,3> fa_flux0,
                           const Accessor<double,3> fa_flux0pi,
                           const double epsi, int *total_converged)
{
  // We know there is never more than 32 warps in a CTA
  __shared__ int trampoline[32];

  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int z = blockIdx.z * blockDim.z + threadIdx.z;
  const Point<3> p = origin + Point<3>(x,y,z);

  const double tolr = 1.0e-12;

  double flux0pi = fa_flux0pi[p];
  double df = 1.0;
  if (fabs(flux0pi) < tolr) {
    flux0pi = 1.0;
    df = 0.0;
  }
  double flux0 = fa_flux0[p];
  df = fabs( (flux0 / flux0pi) - df );
  int local_converged = 1;
  if ((df >= -INFINITY) && (df > epsi))
    local_converged = 0;
  // Perform a local reduction inside the CTA
  // Butterfly reduction across all threads in all warps
  unsigned laneid;
  asm volatile("mov.u32 %0, %laneid;" : "=r"(laneid) : );
  const unsigned warpid = 
    ((threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x) >> 5;
  for (int i = 16; i >= 1; i/=2)
    local_converged += __shfl_xor(local_converged, i, 32);
  // Initialize the trampoline
  if (warpid == 0)
    trampoline[laneid] = 0;
  __syncthreads();
  // First thread in each warp writes out all values
  if (laneid == 0)
    trampoline[warpid] = local_converged;
  __syncthreads();
  // Butterfly reduction across all thread in the first warp
  if (warpid == 0) {
    local_converged = trampoline[laneid];
    for (int i = 16; i >= 1; i/=2)
      local_converged += __shfl_xor(local_converged, i, 32);
    // First thread does the atomic
    if (laneid == 0)
      atomicAdd(total_converged, local_converged);
  }
}

__host__
bool run_inner_convergence(const Rect<3> subgrid_bounds,
                           const std::vector<Accessor<double,3> > &fa_flux0,
                           const std::vector<Accessor<double,3> > &fa_flux0pi,
                           const double epsi)
{
  int *converged_d;
  hipMalloc((void**)&converged_d, sizeof(int));
  // Initialize the result
  hipMemset(converged_d, 0/*value*/, 1/*count*/); 
  // Launch the kernels
  const int x_range = (subgrid_bounds.hi[0] - subgrid_bounds.lo[0]) + 1;
  const int y_range = (subgrid_bounds.hi[1] - subgrid_bounds.lo[1]) + 1;
  const int z_range = (subgrid_bounds.hi[2] - subgrid_bounds.lo[2]) + 1;

  dim3 block(gcd(x_range,32),gcd(y_range,4),gcd(z_range,4));
  dim3 grid(x_range/block.x, y_range/block.y, z_range/block.z);

  assert(fa_flux0.size() == fa_flux0pi.size());
  for (unsigned idx = 0; idx < fa_flux0.size(); idx++) {
    gpu_inner_convergence<<<grid,block>>>(subgrid_bounds.lo,
                                          fa_flux0[idx], fa_flux0pi[idx],
                                          epsi, converged_d); 
  }
  // Copy back: CUDA hijack synchronizes for us
  int converged_h;
  hipMemcpy(&converged_h, converged_d, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(converged_d);
  // We've converged if the total converged points are the number of tests
  return (converged_h == int(x_range * y_range * z_range * fa_flux0.size()));
}

